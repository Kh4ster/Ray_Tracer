#include "hip/hip_runtime.h"
#ifndef __CUDA_SAFE_CALL
#define __CUDA_SAFE_CALL(call) do { hipError_t cuda_error = call; if(cuda_error != hipSuccess) { std::cerr << "CUDA Error: " << hipGetErrorString(cuda_error) << ", " << __FILE__ << ", line " << __LINE__ << std::endl;} } while(0)
#endif

#include "device_atomic_functions.h"
#include "hip/hip_runtime.h"
#include ""	// CUDA includes

#include "Image.hpp"
#include "PerspectiveCamera.hpp"
#include "Vector3.hpp"
#include "ShapeSet.hpp"
#include "Sphere.hpp"
#include "Engine.hpp"
#include "Lambertian.hpp"
#include "Metal.hpp"
#include "Engine.hpp"
#include "Device_color.hpp"
#include "Device_Sphere.hpp"
#include "Device_Intersection.hpp"
#include "Device_Lambertian.hpp"
#include "Device_Metal.hpp"
#include "Device_PerspectiveCamera.hpp"

#include <iostream>
#include <vector>

using Point3 = Vector3; // Points and vectors are basically the same thing + simplifies calculation between them
using Vector2 = Vector3; // To handle vectors on the scene
using Point2 = Vector2; // To handle points on the scene

void init_scene(ShapeSet& scene);
void host_ray_tracer(void);
void device_ray_tracer(void);
__global__
void device_ray_trace(Device_color * image, const unsigned width, const unsigned height, Device_PerspectiveCamera * camera, Device_Sphere * scene, unsigned nb_sphere);

int main(void)
{
	std::cout << "Enter h to execute on host (CPU), d to execute on device (GPU)" << std::endl;
	
	char choice;
	std::cin >> choice;
	
	if (choice == 'h')
		host_ray_tracer();
	else if (choice == 'd')
		device_ray_tracer();

	return 0;
}

static void device_ray_tracer(void)
{
	constexpr size_t width = 1920;
	constexpr size_t height = 1080;
	constexpr float pi = 3.14f;
	constexpr float exposure = 1.0f;
	constexpr float gamma = 2.2f;
	constexpr unsigned nb_spheres = 4;
	constexpr unsigned warp_size = 32;

	Image image(width, height);

	Device_PerspectiveCamera h_camera(Point3(0.0f, 0.0f, 1.0f), Vector3(0.0f, 0.0f, -1.0f), 25.0f * pi / 180.0f, static_cast<float>(width) / static_cast<float>(height));
	Device_Sphere h_spheres[nb_spheres]
	{
		Device_Sphere(Point3(-1.0f, 0.0f, -1.0f), 0.5f, Device_Material::DEVICE_LAMB),
		Device_Sphere(Point3(0.0f, -100.5f, -1.0f), 100.0f, Device_Material::DEVICE_LAMB),
		Device_Sphere(Point3(1.0f, 0.0f, -1.0f), 0.5f, Device_Material::DEVICE_LAMB),
		Device_Sphere(Point3(0.0f, 0.0f, -1.0f), 0.5f, Device_Material::DEVICE_METAL)
	};
	Device_color h_color;
	float h_r[width * height];
	float h_g[width * height];
	float h_b[width * height];
	
	Device_PerspectiveCamera *d_camera;
	Device_Sphere *d_spheres;
	Device_color* d_color;
	float* d_r;
	float* d_g;
	float* d_b;

	dim3 block_size(warp_size, 1, 1);
	dim3 grid_size(width / warp_size + 1, height / warp_size + 1, 1);

	__CUDA_SAFE_CALL(hipMalloc((void**)d_camera, sizeof(Device_PerspectiveCamera)));
	__CUDA_SAFE_CALL(hipMalloc((void**)d_spheres, sizeof(Device_Sphere) * nb_spheres));
	__CUDA_SAFE_CALL(hipMalloc((void**)d_color, sizeof(Device_color)));
	__CUDA_SAFE_CALL(hipMalloc((void**)d_r, sizeof(float) * width * height));
	__CUDA_SAFE_CALL(hipMalloc((void**)d_g, sizeof(float) * width * height));
	__CUDA_SAFE_CALL(hipMalloc((void**)d_b, sizeof(float) * width * height));

	__CUDA_SAFE_CALL(hipMemcpyAsync(d_camera, &h_camera, sizeof(Device_PerspectiveCamera), hipMemcpyHostToDevice));
	__CUDA_SAFE_CALL(hipMemcpyAsync(d_spheres, h_spheres, sizeof(Device_Sphere) * nb_spheres, hipMemcpyHostToDevice));
	__CUDA_SAFE_CALL(hipMemcpyAsync(&(d_color->r), d_r, sizeof(float*), hipMemcpyDeviceToDevice));
	__CUDA_SAFE_CALL(hipMemcpyAsync(&(d_color->g), d_g, sizeof(float*), hipMemcpyDeviceToDevice));
	__CUDA_SAFE_CALL(hipMemcpyAsync(&(d_color->b), d_b, sizeof(float*), hipMemcpyDeviceToDevice));

	device_ray_trace<< <grid_size, block_size >> > (d_color, width, height, d_camera, d_spheres, nb_spheres);
	hipDeviceSynchronize();

	__CUDA_SAFE_CALL(hipMemcpyAsync(d_r, &(d_color->r), sizeof(float*), hipMemcpyDeviceToDevice));
	__CUDA_SAFE_CALL(hipMemcpyAsync(d_g, &(d_color->g), sizeof(float*), hipMemcpyDeviceToDevice));
	__CUDA_SAFE_CALL(hipMemcpyAsync(d_b, &(d_color->b), sizeof(float*), hipMemcpyDeviceToDevice));
	
	__CUDA_SAFE_CALL(hipMemcpyAsync(h_r, d_b, sizeof(float) * width * height, hipMemcpyDeviceToDevice));
	__CUDA_SAFE_CALL(hipMemcpyAsync(h_g, d_r, sizeof(float) * width * height, hipMemcpyDeviceToDevice));
	__CUDA_SAFE_CALL(hipMemcpyAsync(h_b, d_g, sizeof(float) * width * height, hipMemcpyDeviceToDevice));


	image.create_image("out_image.ppm", exposure, gamma);
}

__device__
void device_set_color(unsigned x, unsigned y, unsigned width, Device_color *color, Ray* ray, Device_Sphere* spheres, unsigned nb_sphere, int depth)
{
	bool intersect = false;
	Device_Intersection intersection(ray);
	for (unsigned i = 0; i < nb_sphere; ++i)
	{
		if (spheres[i].intersect(intersection))
		{
			if (spheres[i].material_ == Device_Material::DEVICE_LAMB)
			{
				Ray scattered;
				Device_Lambertian::scatter(intersection, scattered);
				if (depth < 2)
					device_set_color(x, y, width, color, &scattered, spheres, nb_sphere, depth + 1);
				else
					color->set(x, y, 0.0f, 0.0f, 0.0f);
			}
			else
			{
				Ray scattered;
				if (Device_Metal::scatter(intersection, scattered) && depth < 2)
					device_set_color(x, y, width, color, &scattered, spheres, nb_sphere, depth + 1);
				else
					color->set(x, y, 0.0f, 0.0f, 0.0f);
			}
			intersect = true;
		}
	}
	if (!intersect)
		color->set(x, y, 1.0f, 1.0f, 1.0f);
}

__global__
void device_ray_trace(Device_color* image, const unsigned width, const unsigned height, Device_PerspectiveCamera* camera, Device_Sphere* scene, unsigned nb_sphere)
{
	unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned y = blockIdx.y * blockDim.y + threadIdx.y;
	unsigned threadId = y * width + x;

	Point2 screen_coord((2.0f * static_cast<float>(x)) / static_cast<float>(width) - 1.0f,
		(-2.0f * static_cast<float>(y)) / static_cast<float>(height) + 1.0f);

	Ray ray = camera->make_ray(screen_coord);
	device_set_color(x, y, width, image, &ray, scene, nb_sphere, 0);
}

static void host_ray_tracer(void)
{
	constexpr size_t width = 1920;
	constexpr size_t height = 1080;
	constexpr float pi = 3.14f;
	constexpr float exposure = 1.0f;
	constexpr float gamma = 2.2f;

	Image image(width, height);

	PerspectiveCamera camera(Point3(0.0f, 0.0f, 1.0f), Vector3(0.0f, 0.0f, -1.0f), 25.0f * pi / 180.0f, static_cast<float>(width) / static_cast<float>(height));

	Engine engine;

	ShapeSet scene;
	init_scene(scene);

	engine.ray_trace(image, camera, scene);

	image.create_image("out_image.ppm", exposure, gamma);
}

static void init_scene(ShapeSet& scene)
{
	Sphere* red_sphere_left = new Sphere(Point3(-1.0f, 0.0f, -1.0f),
		0.5f,
		new Lambertian(Vector3(0.8f, 0.3f, 0.3f)));
	scene.add_shape(red_sphere_left);

	Sphere* bottom_big_sphere = new Sphere(Point3(0.0f, -100.5f, -1.0f),
		100.0f,
		new Lambertian(Vector3(0.8f, 0.3f, 0.3f)));
	scene.add_shape(bottom_big_sphere);

	Sphere* red_sphere_right = new Sphere(Point3(1.0f, 0.0f, -1.0f),
		0.5f,
		new Lambertian(Vector3(0.8f, 0.3f, 0.3f)));
	scene.add_shape(red_sphere_right);

	Sphere* center_mirror = new Sphere(Point3(0.0f, 0.0f, -1.0f),
		0.5f,
		new Metal(Vector3(0.95f, 0.95f, 0.95f)));
	scene.add_shape(center_mirror);
}